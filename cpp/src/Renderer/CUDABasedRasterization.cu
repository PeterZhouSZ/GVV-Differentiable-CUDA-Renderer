#include "hip/hip_runtime.h"

//==============================================================================================//

#include <hip/hip_runtime.h> 
#include "../Utils/cudaUtil.h"
#include "CUDABasedRasterizationInput.h"
#include "../Utils/CameraUtil.h"
#include "../Utils/IndexHelper.h"
#include "../Utils/cuda_SimpleMatrixUtil.h"
#include "../Utils/RendererUtil.h"

#ifndef FLT_MAX
#define FLT_MAX  1000000
#endif

//==============================================================================================//
//Render buffers
//==============================================================================================//

/*
Initializes all arrays
*/
__global__ void initializeDevice(CUDABasedRasterizationInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx<input.w*input.h*input.numberOfCameras)
	{
		input.d_depthBuffer[idx] = INT_MAX;

		input.d_faceIDBuffer[idx * 4 + 0] = -1;
		input.d_faceIDBuffer[idx * 4 + 1] = -1;
		input.d_faceIDBuffer[idx * 4 + 2] = -1;
		input.d_faceIDBuffer[idx * 4 + 3] = -1;

		input.d_barycentricCoordinatesBuffer[3 * idx + 0] = 0.f;
		input.d_barycentricCoordinatesBuffer[3 * idx + 1] = 0.f;
		input.d_barycentricCoordinatesBuffer[3 * idx + 2] = 0.f;

		input.d_renderBuffer[3 * idx + 0] = 0.f;
		input.d_renderBuffer[3 * idx + 1] = 0.f;
		input.d_renderBuffer[3 * idx + 2] = 0.f;
	}
}

//==============================================================================================//

/*
Project the vertices into the image plane and store depth value
*/
__global__ void projectVerticesDevice(CUDABasedRasterizationInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * input.N)
	{
		int2 index = index1DTo2D(input.numberOfCameras, input.N, idx);
		int idc = index.x;
		int idv = index.y;

		float3 v0 = input.d_vertices[idv];

		float3 c_v0 = getCamSpacePoint(&input.d_cameraExtrinsics[3 * idc], v0);
		float3 i_v0 = projectPointFloat3(&input.d_cameraIntrinsics[3 * idc], c_v0);

		input.d_projectedVertices[idx] = i_v0;
	}
}

//==============================================================================================//

/*
Computes the face normals
*/
__global__ void renderFaceNormalDevice(CUDABasedRasterizationInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * input.F)
	{
		int2 index = index1DTo2D(input.numberOfCameras, input.F, idx);
		int idc = index.x;
		int idf = index.y;

		int indexv0 = input.d_facesVertex[idf].x;
		int indexv1 = input.d_facesVertex[idf].y;
		int indexv2 = input.d_facesVertex[idf].z;

		float3 v0 = input.d_vertices[indexv0];
		float3 v1 = input.d_vertices[indexv1];
		float3 v2 = input.d_vertices[indexv2];

		float3 c_v0 = getCamSpacePoint(&input.d_cameraExtrinsics[3 * idc], v0);
		float3 c_v1 = getCamSpacePoint(&input.d_cameraExtrinsics[3 * idc], v1);
		float3 c_v2 = getCamSpacePoint(&input.d_cameraExtrinsics[3 * idc], v2);

		input.d_faceNormal[idx] = cross(c_v1 - c_v0, c_v2 - c_v0);
	}
}

//==============================================================================================//

/*
Computes the vertex normals
*/
__global__ void renderVertexNormalDevice(CUDABasedRasterizationInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * input.N)
	{
		int2 index = index1DTo2D(input.numberOfCameras, input.N, idx);
		int idv = index.y;

		int2 verFaceId = input.d_vertexFacesId[idv];
		float3 vertNorm;
		for (int i = verFaceId.x; i<verFaceId.x + verFaceId.y; i++)
		{
			int faceId = input.d_vertexFaces[i];

			if (i == verFaceId.x)
				vertNorm = input.d_faceNormal[faceId];
			else
			{
				vertNorm.x = vertNorm.x + input.d_faceNormal[faceId].x;
				vertNorm.y = vertNorm.y + input.d_faceNormal[faceId].y;
				vertNorm.z = vertNorm.z + input.d_faceNormal[faceId].z;
			}
		}
		input.d_vertexNormal[idx] = vertNorm;
	}
}


//==============================================================================================//

/*
Project the vertices into the image plane,
computes the 2D bounding box per triangle in the image plane
and computes the maximum bounding box for all triangles of the mesh
*/
__global__ void projectFacesDevice(CUDABasedRasterizationInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * input.F)
	{
		int2 index = index1DTo2D(input.numberOfCameras, input.F, idx);
		int idc = index.x;
		int idf = index.y;

		int indexv0 = input.d_facesVertex[idf].x;
		int indexv1 = input.d_facesVertex[idf].y;
		int indexv2 = input.d_facesVertex[idf].z;

		float3 i_v0 = input.d_projectedVertices[idc* input.N + indexv0];
		float3 i_v1 = input.d_projectedVertices[idc* input.N + indexv1];
		float3 i_v2 = input.d_projectedVertices[idc* input.N + indexv2];

		input.d_BBoxes[idx].x = fmaxf(fminf(i_v0.x, fminf(i_v1.x, i_v2.x)) - 0.5f, 0);  //minx
		input.d_BBoxes[idx].y = fmaxf(fminf(i_v0.y, fminf(i_v1.y, i_v2.y)) - 0.5f, 0);  //miny

		input.d_BBoxes[idx].z = fminf(fmaxf(i_v0.x, fmaxf(i_v1.x, i_v2.x)) + 0.5f, input.w - 1);   //maxx
		input.d_BBoxes[idx].w = fminf(fmaxf(i_v0.y, fmaxf(i_v1.y, i_v2.y)) + 0.5f, input.h - 1);  //maxy
	}
}

//==============================================================================================//

/*
Render the depth, faceId and barycentricCoordinates buffers
*/
__global__ void renderDepthBufferDevice(CUDABasedRasterizationInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * input.F)
	{
		int2 index = index1DTo2D(input.numberOfCameras, input.F, idx);
		int idc = index.x;
		int idf = index.y;

		int indexv0 = input.d_facesVertex[idf].x;
		int indexv1 = input.d_facesVertex[idf].y;
		int indexv2 = input.d_facesVertex[idf].z;

		float3 vertex0 = input.d_projectedVertices[input.N*idc + indexv0];
		float3 vertex1 = input.d_projectedVertices[input.N*idc + indexv1];
		float3 vertex2 = input.d_projectedVertices[input.N*idc + indexv2];

		for (int u = input.d_BBoxes[idx].x; u <= input.d_BBoxes[idx].z; u++)
		{
			for (int v = input.d_BBoxes[idx].y; v <= input.d_BBoxes[idx].w; v++)
			{
				float2 pixelCenter1 = make_float2(u + 0.5f, v + 0.5f);
				
				float3 abc = uv2barycentric(pixelCenter1.x, pixelCenter1.y, input.d_vertices[indexv0], input.d_vertices[indexv1], input.d_vertices[indexv2], input.d_inverseExtrinsics + idc * 4, input.d_inverseProjection + idc * 4);
				
				float z = FLT_MAX;
				
				bool isInsideTriangle = (abc.x >= -0.001f) && (abc.y >= -0.001f) && (abc.z >= -0.001f) && (abc.x <= 1.001f) && (abc.y <= 1.001f) && (abc.z <= 1.001f);

				if (isInsideTriangle)
				{
					z = 1.f / (abc.x / vertex0.z + abc.y / vertex1.z + abc.z / vertex2.z); //Perspective-Correct Interpolation

					int pixelId = idc* input.w* input.h + input.w * v + u;
					atomicMin(&input.d_depthBuffer[pixelId], z);
				}
			}
		}
	}
}

//==============================================================================================//

/*
Render the faceId and barycentricCoordinates buffers
*/
__global__ void renderBuffersDevice(CUDABasedRasterizationInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * input.F)
	{
		int2 index = index1DTo2D(input.numberOfCameras, input.F, idx);
		int idc = index.x;
		int idf = index.y;

		int indexv0 = input.d_facesVertex[idf].x;
		int indexv1 = input.d_facesVertex[idf].y;
		int indexv2 = input.d_facesVertex[idf].z;

		float3 vertex0 = input.d_projectedVertices[input.N*idc + indexv0];
		float3 vertex1 = input.d_projectedVertices[input.N*idc + indexv1];
		float3 vertex2 = input.d_projectedVertices[input.N*idc + indexv2];

		for (int u = input.d_BBoxes[idx].x; u <= input.d_BBoxes[idx].z; u++)
		{
			for (int v = input.d_BBoxes[idx].y; v <= input.d_BBoxes[idx].w; v++)
			{
				float2 pixelCenter1 = make_float2(u + 0.5f, v + 0.5f);

				float3 abc = uv2barycentric(pixelCenter1.x, pixelCenter1.y, input.d_vertices[indexv0], input.d_vertices[indexv1], input.d_vertices[indexv2], input.d_inverseExtrinsics + idc * 4, input.d_inverseProjection + idc * 4);

				bool isInsideTriangle = (abc.x >= -0.001f) && (abc.y >= -0.001f) && (abc.z >= -0.001f) && (abc.x <= 1.001f) && (abc.y <= 1.001f) && (abc.z <= 1.001f);
	
				float z = 1.f / (abc.x / vertex0.z + abc.y / vertex1.z + abc.z / vertex2.z); //Perspective-Correct Interpolation

				int pixelId = idc* input.w* input.h + input.w * v + u;

				if (isInsideTriangle && (int)z == input.d_depthBuffer[pixelId])
				{
					int pixelId2 = 3 * idc* input.w * input.h + 3 * input.w * v + 3 * u;
					int pixelId3 = 4 * idc* input.w * input.h + 4 * input.w * v + 4 * u;

					//face buffer
					input.d_faceIDBuffer[pixelId3 + 0] = idf;
					input.d_faceIDBuffer[pixelId3 + 1] = indexv0;
					input.d_faceIDBuffer[pixelId3 + 2] = indexv1;
					input.d_faceIDBuffer[pixelId3 + 3] = indexv2;

					//barycentric buffer
					input.d_barycentricCoordinatesBuffer[pixelId2 + 0] = abc.x;
					input.d_barycentricCoordinatesBuffer[pixelId2 + 1] = abc.y;
					input.d_barycentricCoordinatesBuffer[pixelId2 + 2] = abc.z;

					//shading
					float3 v0_norm = input.d_vertexNormal[input.N*idc + indexv0];
					float3 v1_norm = input.d_vertexNormal[input.N*idc + indexv1];
					float3 v2_norm = input.d_vertexNormal[input.N*idc + indexv2];
					float3 pixNorm = v0_norm * abc.x + v1_norm * abc.y + v2_norm * abc.z;
					float pixNormNorm = sqrtf(pixNorm.x*pixNorm.x + pixNorm.y*pixNorm.y + pixNorm.z*pixNorm.z);
					pixNorm = pixNorm / pixNormNorm;

					//render buffer
					if (input.renderMode == RenderMode::Textured)
					{
						float2 texCoord0 = make_float2(input.d_textureCoordinates[idf * 3 * 2 + 0 * 2 + 0], 1.f - input.d_textureCoordinates[idf * 3 * 2 + 0 * 2 + 1]);
						float2 texCoord1 = make_float2(input.d_textureCoordinates[idf * 3 * 2 + 1 * 2 + 0], 1.f - input.d_textureCoordinates[idf * 3 * 2 + 1 * 2 + 1]);
						float2 texCoord2 = make_float2(input.d_textureCoordinates[idf * 3 * 2 + 2 * 2 + 0], 1.f - input.d_textureCoordinates[idf * 3 * 2 + 2 * 2 + 1]);
						float2 finalTexCoord = texCoord0* abc.x + texCoord1* abc.y + texCoord2* abc.z;
						finalTexCoord.x = finalTexCoord.x * input.texWidth;
						finalTexCoord.y = finalTexCoord.y * input.texHeight;

						finalTexCoord.x = fmaxf(finalTexCoord.x, 0);
						finalTexCoord.x = fminf(finalTexCoord.x, input.texWidth - 1);
						finalTexCoord.y = fmaxf(finalTexCoord.y, 0);
						finalTexCoord.y = fminf(finalTexCoord.y, input.texHeight - 1);

						float3 color = make_float3(input.d_textureMap[3 * input.texWidth *(int)finalTexCoord.y + 3 * (int)finalTexCoord.x + 0],
							input.d_textureMap[3 * input.texWidth *(int)finalTexCoord.y + 3 * (int)finalTexCoord.x + 1],
							input.d_textureMap[3 * input.texWidth *(int)finalTexCoord.y + 3 * (int)finalTexCoord.x + 2]);

						float3 colorShaded = getShading(color, pixNorm, input.d_shCoeff + (idc * 27));
						input.d_renderBuffer[pixelId2 + 0] = colorShaded.x;
						input.d_renderBuffer[pixelId2 + 1] = colorShaded.y;
						input.d_renderBuffer[pixelId2 + 2] = colorShaded.z;
					}
					else if (input.renderMode == RenderMode::VertexColor)
					{
						//vertex color buffer
						float3 color = make_float3(
							input.d_vertexColor[indexv0].x * abc.x + input.d_vertexColor[indexv1].x * abc.y + input.d_vertexColor[indexv2].x * abc.z,
							input.d_vertexColor[indexv0].y * abc.x + input.d_vertexColor[indexv1].y * abc.y + input.d_vertexColor[indexv2].y * abc.z,
							input.d_vertexColor[indexv0].z * abc.x + input.d_vertexColor[indexv1].z * abc.y + input.d_vertexColor[indexv2].z * abc.z);

						float3 colorShaded = getShading(color, pixNorm, input.d_shCoeff + (idc * 27));
						input.d_renderBuffer[pixelId2 + 0] = colorShaded.x; 
						input.d_renderBuffer[pixelId2 + 1] = colorShaded.y; 
						input.d_renderBuffer[pixelId2 + 2] = colorShaded.z;
					}
				}
			}
		}
	}
}

//==============================================================================================//

extern "C" void renderBuffersGPU(CUDABasedRasterizationInput& input)
{
	initializeDevice			<< <(input.w*input.h*input.numberOfCameras + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> > (input);

	projectVerticesDevice		<< <(input.N*input.numberOfCameras + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >(input);

	projectFacesDevice			<< <(input.F*input.numberOfCameras + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >(input);

	renderFaceNormalDevice		<< <(input.F*input.numberOfCameras + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >(input);

	renderVertexNormalDevice	<< <(input.N*input.numberOfCameras + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >(input);

	renderDepthBufferDevice		<< <(input.F*input.numberOfCameras + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >(input);

	renderBuffersDevice			<< <(input.F*input.numberOfCameras + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >(input);
}
