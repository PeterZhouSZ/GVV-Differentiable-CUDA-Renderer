#include "hip/hip_runtime.h"

//==============================================================================================//

#include <hip/hip_runtime.h> 
#include "../Utils/cudaUtil.h"
#include "../Utils/cuda_SimpleMatrixUtil.h"
#include "../Utils/RendererUtil.h"
#include "CUDABasedRasterizationGradInput.h"
#include "../Utils/CameraUtil.h"
#include "../Utils/IndexHelper.h"

//==============================================================================================//

/*
Initialize gradients for lighting 
*/
__global__ void initBuffersGradDevice2(CUDABasedRasterizationGradInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * 27)
	{
		input.d_shCoeffGrad[idx] = 0.f;
	}
}

//==============================================================================================//

/*
Initialize gradients for texture
*/
__global__ void initBuffersGradDevice1(CUDABasedRasterizationGradInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.texHeight * input.texWidth)
	{
		input.d_textureGrad[idx] = make_float3(0.f,0.f,0.f);
	}
}

//==============================================================================================//

/*
Initialize gradients for mesh pos and color
*/
__global__ void initBuffersGradDevice0(CUDABasedRasterizationGradInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.N)
	{
		input.d_vertexPosGrad[idx]	 = make_float3(0.f, 0.f, 0.f);
		input.d_vertexColorGrad[idx] = make_float3(0.f, 0.f, 0.f);
	}
}

//==============================================================================================//

/*
Get gradients for vertex color buffer
*/
__global__ void renderBuffersGradDevice(CUDABasedRasterizationGradInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * input.w * input.h)
	{
		////////////////////////////////////////////////////////////////////////
		//INDEXING
		////////////////////////////////////////////////////////////////////////

		int3 index = index1DTo3D(input.numberOfCameras, input.h, input.w, idx);
		int idc = index.x;
		int idh = index.y;
		int idw = index.z;
		int idf = input.d_faceIDBuffer[idx];

		if (idf == -1)
			return;

		////////////////////////////////////////////////////////////////////////
		//INIT
		////////////////////////////////////////////////////////////////////////

		float2 bccTmp	= input.d_barycentricCoordinatesBuffer[idx];
		float3 bcc		= make_float3(bccTmp.x, bccTmp.y, 1.f - bccTmp.x - bccTmp.y);

		int3   faceVerticesIds  = input.d_facesVertex[idf];
		const float* shCoeff	= input.d_shCoeff + idc * 27;

		float3 vertexPos0 = input.d_vertices[faceVerticesIds.x];
		float3 vertexPos1 = input.d_vertices[faceVerticesIds.y];
		float3 vertexPos2 = input.d_vertices[faceVerticesIds.z];
		float3 vertexCol0 = input.d_vertexColor[faceVerticesIds.x];
		float3 vertexCol1 = input.d_vertexColor[faceVerticesIds.y];
		float3 vertexCol2 = input.d_vertexColor[faceVerticesIds.z];
		float3 vertexNor0 = input.d_vertexNormal[idc*input.N + faceVerticesIds.x];
		float3 vertexNor1 = input.d_vertexNormal[idc*input.N + faceVerticesIds.y];
		float3 vertexNor2 = input.d_vertexNormal[idc*input.N + faceVerticesIds.z];
		float2 texCoord0 = make_float2(input.d_textureCoordinates[idf * 3 * 2 + 0 * 2 + 0], 1.f - input.d_textureCoordinates[idf * 3 * 2 + 0 * 2 + 1]);
		float2 texCoord1 = make_float2(input.d_textureCoordinates[idf * 3 * 2 + 1 * 2 + 0], 1.f - input.d_textureCoordinates[idf * 3 * 2 + 1 * 2 + 1]);
		float2 texCoord2 = make_float2(input.d_textureCoordinates[idf * 3 * 2 + 2 * 2 + 0], 1.f - input.d_textureCoordinates[idf * 3 * 2 + 2 * 2 + 1]);

		float3 pixNormUn	= bcc.x * vertexNor0 + bcc.y * vertexNor1 + bcc.z * vertexNor2;
		float  pixNormVal	= sqrtf(pixNormUn.x*pixNormUn.x + pixNormUn.y*pixNormUn.y + pixNormUn.z*pixNormUn.z);
		float3 pixNorm = pixNormUn / pixNormVal;

		////////////////////////////////////////////////////////////////////////
		//VERTEX COLOR AND TEXTURE GRAD
		////////////////////////////////////////////////////////////////////////

		float3 pixLight = getIllum(pixNorm, shCoeff);
		mat3x3 JCoAl;
		getJCoAl(JCoAl, pixLight); 

		mat1x3 GVCBVertexColor;
		GVCBVertexColor(0, 0) = input.d_renderBufferGrad[idx].x;
		GVCBVertexColor(0, 1) = input.d_renderBufferGrad[idx].y;
		GVCBVertexColor(0, 2) = input.d_renderBufferGrad[idx].z;

		float2 finalTexCoord = make_float2(0.f, 0.f);

		if (input.renderMode == RenderMode::VertexColor)
		{
			mat3x9 JAlVc;
			getJAlVc(JAlVc, bcc);

			mat1x9 gradVerCol = GVCBVertexColor * JCoAl * JAlVc;

			addGradients9I(gradVerCol.getTranspose(), input.d_vertexColorGrad, faceVerticesIds);
		}
		else if (input.renderMode == RenderMode::Textured)
		{
			mat1x3 gradTexColor = GVCBVertexColor * JCoAl ;

			finalTexCoord = texCoord0* bcc.x + texCoord1* bcc.y + texCoord2* bcc.z;
			finalTexCoord.x = finalTexCoord.x * input.texWidth;
			finalTexCoord.y = finalTexCoord.y * input.texHeight;
			finalTexCoord.x = fmaxf(finalTexCoord.x, 0);
			finalTexCoord.x = fminf(finalTexCoord.x, input.texWidth - 1);
			finalTexCoord.y = fmaxf(finalTexCoord.y, 0);
			finalTexCoord.y = fminf(finalTexCoord.y, input.texHeight - 1);

			atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, finalTexCoord.y, finalTexCoord.x)].x, gradTexColor(0, 0));
			atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, finalTexCoord.y, finalTexCoord.x)].y, gradTexColor(0, 1));
			atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, finalTexCoord.y, finalTexCoord.x)].z, gradTexColor(0, 2));
		}
		else
		{
			printf("Unsupported color mode in renderer gradient! \n");
		}

		////////////////////////////////////////////////////////////////////////
		//LIGHTING GRAD
		////////////////////////////////////////////////////////////////////////

		mat1x3 GVCBLight;
		GVCBLight(0, 0) = input.d_renderBufferGrad[idx].x;
		GVCBLight(0, 1) = input.d_renderBufferGrad[idx].y;
		GVCBLight(0, 2) = input.d_renderBufferGrad[idx].z;

		mat3x3 JCoLi;
		float3 pixAlb = make_float3(0.f, 0.f, 0.f);
		if (input.renderMode == RenderMode::VertexColor)
		{
			pixAlb = bcc.x * vertexCol0 + bcc.y * vertexCol1 + bcc.z * vertexCol2;
		}
		else if (input.renderMode == RenderMode::Textured)
		{
			pixAlb = make_float3(input.d_textureMap[index3DTo1D(input.texHeight, input.texWidth, 3, finalTexCoord.y, finalTexCoord.x, 0)],
								 input.d_textureMap[index3DTo1D(input.texHeight, input.texWidth, 3, finalTexCoord.y, finalTexCoord.x, 1)],
								 input.d_textureMap[index3DTo1D(input.texHeight, input.texWidth, 3, finalTexCoord.y, finalTexCoord.x, 2)]);
		}
		getJCoLi(JCoLi, pixAlb);

		mat3x9 JLiGmR;
		getJLiGm(JLiGmR, 0, pixNorm);
		mat3x9 JLiGmG;
		getJLiGm(JLiGmG, 1, pixNorm);
		mat3x9 JLiGmB;
		getJLiGm(JLiGmB, 2, pixNorm);

		mat1x9 gradSHCoeffR = GVCBLight * JCoLi * JLiGmR;
		mat1x9 gradSHCoeffG = GVCBLight * JCoLi * JLiGmG;
		mat1x9 gradSHCoeffB = GVCBLight * JCoLi * JLiGmB;

		addGradients9(gradSHCoeffR, &input.d_shCoeffGrad[idc * 27     ]);
		addGradients9(gradSHCoeffG, &input.d_shCoeffGrad[idc * 27 +  9]);
		addGradients9(gradSHCoeffB, &input.d_shCoeffGrad[idc * 27 + 18]);

		////////////////////////////////////////////////////////////////////////
		//VERTEX POS GRAD
		////////////////////////////////////////////////////////////////////////

		mat1x3 GVCBPosition;
		GVCBPosition(0, 0) = input.d_renderBufferGrad[idx].x;
		GVCBPosition(0, 1) = input.d_renderBufferGrad[idx].y;
		GVCBPosition(0, 2) = input.d_renderBufferGrad[idx].z;

		mat3x3 JNoNu;
		getJNoNu(JNoNu, pixNormUn, pixNormVal);

		mat3x3 JLiNo;
		getJLiNo(JLiNo, pixNorm, shCoeff);

		mat3x3 TR = getRotationMatrix(&input.d_cameraExtrinsics[3 * idc]);

		/////////////////////

		mat3x3 JAlBc;

		if (input.renderMode == RenderMode::VertexColor)
		{
			getJAlBc(JAlBc, vertexCol0, vertexCol1, vertexCol2);
		}
		else if (input.renderMode == RenderMode::Textured)
		{
			getJAlTexBc(JAlBc, input.d_textureMap, finalTexCoord, texCoord0, texCoord1, texCoord2, input.texWidth, input.texHeight);
		}

		mat3x3 JNoBc;
		getJNoBc(JNoBc, vertexNor0, vertexNor1, vertexNor2);
		
		mat3x9 JBcVp;
		float3 o = make_float3(0.f, 0.f, 0.f);
		float3 d = make_float3(0.f, 0.f, 0.f);
		float2 pixelPos = make_float2(idw, idh);
		getRayCuda2(pixelPos, o, d, input.d_inverseExtrinsics + idc * 4, input.d_inverseProjection + idc * 4);
		
		dJBCDVerpos(JBcVp,o,d,vertexPos0, vertexPos1, vertexPos2);

		mat1x9 gradVerPos = GVCBPosition * JCoAl * JAlBc * JBcVp + GVCBPosition * JCoLi * JLiNo * JNoNu * JNoBc * JBcVp;

		addGradients9I(gradVerPos.getTranspose(), input.d_vertexPosGrad, faceVerticesIds);

		////////////////////

		for (int i = 0; i < 3; i++)
		{
			mat3x3 JNuNvx;
			JNuNvx.setIdentity();
			int idv = -1;

			//
			if (i == 0) 
			{ 
				idv = faceVerticesIds.x; 
				JNuNvx = bcc.x * JNuNvx; 
			}
			else if (i == 1) 
			{ 
				idv = faceVerticesIds.y; 
				JNuNvx = bcc.y * JNuNvx; 
			}
			else 
			{ 
				idv = faceVerticesIds.z; 
				JNuNvx = bcc.z * JNuNvx; 
			}

			//
			int2 verFaceId = input.d_vertexFacesId[idv];

			//
			for (int j = verFaceId.x; j < verFaceId.x + verFaceId.y; j++)
			{
				int faceId = input.d_vertexFaces[j];
			
				int3 v_index_inner = input.d_facesVertex[faceId];
				mat3x1 vi = TR * (mat3x1)input.d_vertices[v_index_inner.x];
				mat3x1 vj = TR * (mat3x1)input.d_vertices[v_index_inner.y];
				mat3x1 vk = TR * (mat3x1)input.d_vertices[v_index_inner.z];

				mat3x3 J;
				
				// gradients vi
				getJ_vi(J, TR, vk, vj, vi);
				mat1x3 gradVi = GVCBPosition * JCoLi * JLiNo * JNoNu * JNuNvx * J;
				addGradients(gradVi, &input.d_vertexPosGrad[v_index_inner.x]);

				// gradients vj
				getJ_vj(J, TR, vk, vi);
				mat1x3 gradVj = GVCBPosition * JCoLi * JLiNo * JNoNu * JNuNvx * J;
				addGradients(gradVj, &input.d_vertexPosGrad[v_index_inner.y]);

				// gradients vk
				getJ_vk(J, TR, vj, vi);
				mat1x3 gradVk = GVCBPosition * JCoLi * JLiNo * JNoNu * JNuNvx * J;
				addGradients(gradVk, &input.d_vertexPosGrad[v_index_inner.z]);	
			}
		}
	}
}

//==============================================================================================//

/*
Call to the devices for computing the gradients
*/
extern "C" void renderBuffersGradGPU(CUDABasedRasterizationGradInput& input)
{
	initBuffersGradDevice2    << < (input.numberOfCameras * 27 + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >				(input);

	initBuffersGradDevice1    << < (input.texHeight * input.texWidth + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >		(input);

	initBuffersGradDevice0    << < (input.N + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >								(input);

	renderBuffersGradDevice   << < (input.numberOfCameras*input.w*input.h + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >	(input);
}

//==============================================================================================//


